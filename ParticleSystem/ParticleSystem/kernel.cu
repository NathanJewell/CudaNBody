#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>

//defining program attributes
static const int N = 100;
static const size_t size = N * sizeof(int);
static const int TPB = 512;	//threads per block

__global__ void ARR_ADD(int* res, int* in1, int *in2, int n)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n)
		res[index] = in1[index] + in2[index];
}

int main()
{



	//allocating host memory
	int *res = (int*) malloc(size);
	int *in1 = (int*) malloc(size);
	int *in2 = (int*) malloc(size);

	//defining pointers to device memory 
	int *d_res, *d_in1, *d_in2;

	hipMalloc(&d_res, size);
	hipMalloc(&d_in1, size);
	hipMalloc(&d_in2, size);

	//initializing values
	for (int i = 0; i < N; i++)
	{
		in1[i] = i + 1;
		in2[i] = i + 2;
	}


	//copying to device memory
	hipMemcpy(d_res, res, size, hipMemcpyHostToDevice);
	hipMemcpy(d_in1, in1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_in2, in2, size, hipMemcpyHostToDevice);

	//call function to execute on device
	ARR_ADD << <N/TPB, TPB>> >(d_res, d_in1, d_in2, N);

	//copy result back to host
	hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);

	//free memory on device
	hipFree(d_res);
	hipFree(d_in1);
	hipFree(d_in2);

	//print result to window
	long int checksum = 0;
	for (int i = 0; i < N; i++)
	{
		checksum += res[i];
	}
	std::cout << "Parrelelized N=" << N << " : " << checksum << std::endl;

	//free host memory
	delete res;
	delete in1;
	delete in2;

	int resN[N], in1N[N], in2N[N];
	for (int i = 0; i < N; i++)
	{
		in1N[i] = i + 1;
		in2N[i] = i + 2;
	}
	for (int i = 0; i < N; i++)
	{
		resN[i] = in1N[i] + in2N[i];
	}
	checksum = 0;
	for (int i = 0; i < N; i++)
	{
		checksum += resN[i];
	}
	std::cout << "Standard N=" << N << " : " << checksum << std::endl;

}
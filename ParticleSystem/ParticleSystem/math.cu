#include "hip/hip_runtime.h"
#include "math.cuh"


__device__ float fInvSqrt_D(const float& in)
{
	long i;
	float x2, y;
	const float threehalfs = 1.5F;

	x2 = in * 0.5F;
	y = in;
	i = *(long *)&y;
	i = 0x5f3759df - (i >> 1);
	y = *(float *)&i;
	y = y * (threehalfs - (x2 * y * y));
	y = y * (threehalfs - (x2 * y * y));	//extra precision
	return y;
}


__device__ void doParticle(p_type* pos, p_type* vel, p_type* acc, p_type* mass, int numParticles, int pIndex2, int index2, int thisIndex)
{
	int index = thisIndex;
	int pIndex1 = index * 3;
	//printf("index1 %d \n", index);
	//printf("index2 %d \n", index2);
	if (pIndex1 != pIndex2 && index < numParticles)
	{

		p_type diffx = (pos[pIndex1] - pos[pIndex2]);			//calculating difference between points
		p_type diffy = (pos[pIndex1 + 1] - pos[pIndex2 + 1]);
		p_type diffz = (pos[pIndex1 + 2] - pos[pIndex2 + 2]);

		p_type distsqr = diffx*diffx + diffy*diffy + diffz*diffz;

		if (distsqr < 0)
		{
			distsqr *= -1;
		}

		if (distsqr > -.01 && distsqr < .01)	//want to prevent errors and simulate collision
		{
			//add mass to other particle
			mass[index2] += mass[index];
			mass[index] = 0;

			//move it out of view
			pos[pIndex1] = 100000;
			pos[pIndex1 + 1] = 100000;
			pos[pIndex1 + 2] = 100000;
		}
		else
		{

			p_type attraction = (mass[index2] * mass[index]) / (distsqr * 1000000000000000000);	//gravity equation

			p_type invsqrt = fInvSqrt_D((float)distsqr);
			p_type normx = invsqrt*diffx;
			p_type normy = invsqrt*diffy;
			p_type normz = invsqrt*diffz;

			p_type forcex = normx * -attraction;
			p_type forcey = normy * -attraction;
			p_type forcez = normz * -attraction;

			acc[pIndex1] += forcex;
			acc[pIndex1 + 1] += forcey;
			acc[pIndex1 + 2] += forcez;
		}

	}
}

__global__ void beginFrame(p_type* pos, p_type* vel, p_type* acc, p_type* mass, int numParticles, int numBlocks)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int pIndex1 = index * 3;

	if (index < numParticles)
	{
		for (int i = 0; i < numParticles; i++)
		{
			doParticle(pos, vel, acc, mass, numParticles, pIndex1, index, i);
		}
		//pos[index] = 0;
	}

}


__global__ void ARR_ADD(p_type* getter, const p_type *giver, int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < N)
	{
		getter[index] = getter[index] + giver[index];
	}

}

__global__ void ARR_ADDC(float* result, float* in1, float* in2, int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < N)
	{
		result[index] = in1[index] + in2[index];
	}
}

__global__ void ARR_SET(p_type* getter, const p_type value, int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < N)
	{
		getter[index] = value;
	}
}

__host__ void doFrame(p_type* d_pos, p_type* d_vel, p_type* d_acc, p_type* d_mass, int numParticles, int numBlocks, int numBlocks2)
{
	beginFrame << <numBlocks, TPB >> >(d_pos, d_vel, d_acc, d_mass, numParticles, numBlocks);
	hipError_t err;

	ARR_ADD << <numBlocks2, TPB >> >(d_vel, d_acc, numParticles * 3);

	//p_type* test;
	//test = (p_type*)malloc(sizeof(p_type) * 3 * numParticles);
	//hipMemcpy(test, d_pos, sizeof(p_type) * 3 * numParticles, hipMemcpyDeviceToHost);
	//hipMemcpy(test, d_vel, sizeof(p_type) * 3 * numParticles, hipMemcpyDeviceToHost);
	//hipMemcpy(test, d_acc, sizeof(p_type) * 3 * numParticles, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	ARR_SET << <numBlocks2, TPB >> >(d_acc, 0.0f, numParticles * 3);

	//hipMemcpy(test, d_pos, sizeof(p_type) * 3 * numParticles, hipMemcpyDeviceToHost);
	//hipMemcpy(test, d_vel, sizeof(p_type) * 3 * numParticles, hipMemcpyDeviceToHost);
	//hipMemcpy(test, d_acc, sizeof(p_type) * 3 * numParticles, hipMemcpyDeviceToHost);

	ARR_ADD << <numBlocks2, TPB >> >(d_pos, d_vel, numParticles * 3);

	//hipMemcpy(test, d_pos, sizeof(p_type) * 3 * numParticles, hipMemcpyDeviceToHost);
	//hipMemcpy(test, d_vel, sizeof(p_type) * 3 * numParticles, hipMemcpyDeviceToHost);
	//hipMemcpy(test, d_acc, sizeof(p_type) * 3 * numParticles, hipMemcpyDeviceToHost);

	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}